
#include <hip/hip_runtime.h>
#ifdef OMPD_SUPPORT
#include "ompd-specific.h"
#include "omptarget-nvptx.h"
/**
   * Declaration of symbols to hold struct size and member offset information
    */

__device__ __shared__ static int ompd_target_initialized;

#define ompd_target_declare_access(t,m) __device__ __shared__ uint64_t ompd_access__##t##__##m##_;
OMPD_FOREACH_ACCESS(ompd_target_declare_access)
#undef ompd_target_declare_access

#define ompd_target_declare_sizeof_member(t,m) __device__ __shared__ uint64_t ompd_sizeof__##t##__##m##_;
    OMPD_FOREACH_ACCESS(ompd_target_declare_sizeof_member)
#undef ompd_target_declare_sizeof_member

#define ompd_target_declare_sizeof(t) __device__ __shared__ uint64_t ompd_sizeof__##t##_;
    OMPD_FOREACH_SIZEOF(ompd_target_declare_sizeof)
#undef ompd_target_declare_sizeof

__device__ __shared__
  uint64_t ompd_access__omptarget_nvptx_TaskDescr__items__threadsInTeam_;

__device__ __shared__
  uint64_t ompd_sizeof__omptarget_nvptx_TaskDescr__items__threadsInTeam_;


__device__ void ompd_init ( void )
{
  if (ompd_target_initialized)
    return;

#define ompd_target_init_access(t,m) ompd_access__##t##__##m##_ = (uint64_t)&(((t*)0)->m);
  OMPD_FOREACH_ACCESS(ompd_target_init_access)
#undef ompd_target_init_access

  ompd_access__omptarget_nvptx_TaskDescr__items__threadsInTeam_ =
          (uint64_t)&(((omptarget_nvptx_TaskDescr*)0)->items.threadsInTeam);

#define ompd_target_init_sizeof_member(t,m) ompd_sizeof__##t##__##m##_ = sizeof(((t*)0)->m);
  OMPD_FOREACH_ACCESS(ompd_target_init_sizeof_member)
#undef ompd_target_init_sizeof_member

  ompd_sizeof__omptarget_nvptx_TaskDescr__items__threadsInTeam_ =
    (uint64_t)sizeof(((omptarget_nvptx_TaskDescr*)0)->items.threadsInTeam);

#define ompd_target_init_sizeof(t) ompd_sizeof__##t##_ = sizeof(t);
  OMPD_FOREACH_SIZEOF(ompd_target_init_sizeof)
#undef ompd_target_init_sizeof

  omptarget_nvptx_threadPrivateContext->ompd_levelZeroParallelInfo.level = 0;
  if (isSPMDMode()) {
    omptarget_nvptx_threadPrivateContext->teamContext.levelZeroTaskDescr
        .ompd_thread_info.enclosed_parallel.parallel_tasks =
            &omptarget_nvptx_threadPrivateContext->levelOneTaskDescr[0];
  } else {
    // generic mode
    omptarget_nvptx_threadPrivateContext->ompd_levelZeroParallelInfo
        .parallel_tasks = &omptarget_nvptx_threadPrivateContext->teamContext
            .levelZeroTaskDescr;
  }

  ompd_target_initialized = 1;
}

INLINE void ompd_init_thread(omptarget_nvptx_TaskDescr *currTaskDescr) {
  currTaskDescr->ompd_thread_info.blockIdx_x = blockIdx.x;
  currTaskDescr->ompd_thread_info.threadIdx_x = threadIdx.x;
}

__device__ void ompd_set_device_specific_thread_state(
    omptarget_nvptx_TaskDescr *taskDescr, omp_state_t state) {
    taskDescr->ompd_thread_info.state = state;
}

__device__ void  ompd_set_device_thread_state(omp_state_t state) {
  ompd_set_device_specific_thread_state(getMyTopTaskDescriptor(), state);
}

__device__ void ompd_init_thread_parallel() {
  omptarget_nvptx_TaskDescr *currTaskDescr = getMyTopTaskDescriptor();
  ompd_init_thread(currTaskDescr);
  ompd_set_device_specific_thread_state(currTaskDescr, omp_state_work_parallel);
}

__device__ void ompd_init_thread_master() {
  omptarget_nvptx_TaskDescr *currTaskDescr = getMyTopTaskDescriptor();
  ompd_init_thread(currTaskDescr);
  ompd_set_device_specific_thread_state(currTaskDescr, omp_state_work_serial);
}

__device__ void ompd_bp_parallel_begin (){ asm (""); }
__device__ void ompd_bp_parallel_end (){ asm (""); }
__device__ void ompd_bp_task_begin (){ asm (""); }
__device__ void ompd_bp_task_end (){ asm (""); }
#endif /* OMPD_SUPPORT */
